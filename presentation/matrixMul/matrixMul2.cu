
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#define N 2048 // 矩阵大小 N x N

// CUDA 核函数：执行矩阵乘法
#define TILE_SIZE 32  // 每个线程块的大小

__global__ void matrixMulOptimized(int *A, int *B, int *C, int n) {
    __shared__ int tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ int tile_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    int sum = 0;

    // 遍历子矩阵块
    for (int k = 0; k < n / TILE_SIZE; k++) {
        // **线程块中的线程共同加载 A 和 B 的数据到共享内存**
        tile_A[threadIdx.y][threadIdx.x] = A[row * n + (k * TILE_SIZE + threadIdx.x)];
        tile_B[threadIdx.y][threadIdx.x] = B[(k * TILE_SIZE + threadIdx.y) * n + col];

        __syncthreads(); // **同步，确保所有线程都加载完毕**

        // **计算该线程的结果**
        for (int i = 0; i < TILE_SIZE; i++) {
            sum += tile_A[threadIdx.y][i] * tile_B[i][threadIdx.x];
        }

        __syncthreads(); // **同步，确保所有线程都用完当前共享内存**
    }

    // **写入结果**
    if (row < n && col < n) {
        C[row * n + col] = sum;
    }
}


int main()
{
    int size = N * N * sizeof(int);
    int *h_A  = (int*)malloc(size);// 示例矩阵 A
    int *h_B = (int*)malloc(size); // 示例矩阵 B
    int *h_C = (int*)malloc(size); // 结果矩阵 C
    for (int i = 0; i < N * N; i++)
    {
        h_A[i] = 1;
        h_B[i] = 1;
    }

    int *d_A, *d_B, *d_C; // 显卡里的
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); // 拷贝到GPU中
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32); // 每个线程块有16乘16个线程
    dim3 blocksPerGrid((N + 31) / 32, (N + 31) / 32);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipEventQuery(start);

    matrixMulOptimized<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // 结束的模板
    hipDeviceSynchronize();
    hipEventRecord(stop); 
    hipEventSynchronize(stop); // 相当于清空缓冲器
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop); // 算出时间差
    printf("GPU:Time = %g ms \n", elapsed_time);
    // 清空内存
    hipEventDestroy(start); 
    hipEventDestroy(stop);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
