
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#define N 2048 // 矩阵大小 N x N

// CUDA 核函数：执行矩阵乘法
__global__ void matrixMul(int *A, int *B, int *C, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    for (int k = 0; k < n; k++)
    {
        sum += A[row * n + k] * B[k * n + col];
    }
    C[row * n + col] = sum;
}

int main()
{
    int size = N * N * sizeof(int);
    int *h_A = (int *)malloc(size); // 示例矩阵 A
    int *h_B = (int *)malloc(size); // 示例矩阵 B
    int *h_C = (int *)malloc(size); // 结果矩阵 C
    for (int i = 0; i < N * N; i++)
    {
        h_A[i] = 1;
        h_B[i] = 1;
    }

    int *d_A, *d_B, *d_C; // 显卡里的
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); // 拷贝到GPU中
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32); // 每个线程块有32乘32 个线程
    dim3 blocksPerGrid((N + 31) / 32, (N + 31) / 32);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipEventQuery(start);

    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // 结束的模板
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop); // 相当于清空缓冲器
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop); // 算出时间差
    printf("GPU：Time = %g ms \n", elapsed_time);
    // 清空内存
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
