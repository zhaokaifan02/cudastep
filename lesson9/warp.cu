#include "hip/hip_runtime.h"
#include "error.cuh"
#include <stdio.h>

const unsigned WIDTH = 8;
const unsigned BLOCK_SIZE = 16;
const unsigned FULL_MASK = 0xffffffff; 

__global__ void  test_warp_primitives(void);

int main(int argc, char **argv)
{
    test_warp_primitives<<<1, BLOCK_SIZE>>>();
    CHECK(hipDeviceSynchronize()); //用printf时必要的
    return 0;
}

__global__ void  test_warp_primitives(void)
{
    int tid = threadIdx.x; //线程id
    int lane_id = tid % WIDTH; //块内id
    //WIDTH理解为一种逻辑warp分割，真实的warp永远是32不可改的，但是我们可以人为调整来理解代码
    if (tid == 0) printf("threadIdx.x: ");
    printf("%2d ", tid); 
    if (tid == 0) printf("\n");

    if (tid == 0) printf("lane_id:     ");
    printf("%2d ", lane_id);
    if (tid == 0) printf("\n");

    unsigned mask1 = __ballot_sync(FULL_MASK, tid > 0); //参与的同一个warp内，标记所有tid>0的
    unsigned mask2 = __ballot_sync(FULL_MASK, tid == 0);//标记tid等于0的

    if (tid == 0) printf("FULL_MASK = %x\n", FULL_MASK);
    if (tid == 1) printf("mask1     = %x\n", mask1);
    if (tid == 0) printf("mask2     = %x\n", mask2);

    int result = __all_sync(FULL_MASK, tid); //如果所有参与现成的tid都不为0时才返回1
    if (tid == 0) printf("all_sync (FULL_MASK): %d\n", result);

    result = __all_sync(mask1, tid); //参与线程都不为0才返回1
    if (tid == 1) printf("all_sync     (mask1): %d\n", result);

    result = __any_sync(FULL_MASK, tid); //参与线程有一个不都为0时才返回1
    if (tid == 0) printf("any_sync (FULL_MASK): %d\n", result);

    result = __any_sync(mask2, tid);  //参与线程有一个不都为0时才返回1
    if (tid == 0) printf("any_sync     (mask2): %d\n", result);

    int value = __shfl_sync(FULL_MASK, tid, 2, WIDTH); //获得束内编号为2的线程的tid
    if (tid == 0) printf("shfl:      ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_up_sync(FULL_MASK, tid, 1, WIDTH); //获得t-1现成的tid值
    if (tid == 0) printf("shfl_up:   ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_down_sync(FULL_MASK, tid, 1, WIDTH); //获得t+1线程的tid值
    if (tid == 0) printf("shfl_down: ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");

    value = __shfl_xor_sync(FULL_MASK, tid, 1, WIDTH); //
    if (tid == 0) printf("shfl_xor:  ");
    printf("%2d ", value);
    if (tid == 0) printf("\n");
}
