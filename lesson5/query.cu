
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc,char* argv[1])
{
    int device_id = 0; //要看的GPU编号
    if(argc>1) device_id = atoi(argv[1]); //也可以在调用时指定要看的GPU位置

    hipSetDevice(device_id); //设置gup编号

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);

     printf("Device id:                                 %d\n",
        device_id);
    printf("Device name:                               %s\n",
        prop.name);
    printf("Compute capability:                        %d.%d\n",
        prop.major, prop.minor);
    printf("Amount of global memory:                   %g GB\n",
        prop.totalGlobalMem / (1024.0 * 1024 * 1024));
    printf("Amount of constant memory:                 %g KB\n",
        prop.totalConstMem  / 1024.0);
    printf("Maximum grid size:                         %d %d %d\n",
        prop.maxGridSize[0], 
        prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("Maximum block size:                        %d %d %d\n",
        prop.maxThreadsDim[0], prop.maxThreadsDim[1], 
        prop.maxThreadsDim[2]);
    printf("Number of SMs:                             %d\n",
        prop.multiProcessorCount);
    printf("Maximum amount of shared memory per block: %g KB\n",
        prop.sharedMemPerBlock / 1024.0);
    printf("Maximum amount of shared memory per SM:    %g KB\n",
        prop.sharedMemPerMultiprocessor / 1024.0);
    printf("Maximum number of registers per block:     %d K\n",
        prop.regsPerBlock / 1024);
    printf("Maximum number of registers per SM:        %d K\n",
        prop.regsPerMultiprocessor / 1024);
    printf("Maximum number of threads per block:       %d\n",
        prop.maxThreadsPerBlock);
    printf("Maximum number of threads per SM:          %d\n",
        prop.maxThreadsPerMultiProcessor);




}