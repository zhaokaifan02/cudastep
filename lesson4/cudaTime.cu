#include "error.cuh"
#include <hip/hip_runtime.h>
//cuda有自己的及时方式 cuda event
int main()
{
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start); 
    hipEventQuery(start);

    //需要及时的代码模块

    //结束的模板
    hipEventRecord(stop);
    hipEventSynchronize(stop); //相当于清空缓冲器
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time,start,stop); //算出时间差
    printf("Time = %g ms \n", elapsed_time);
    //清空内存
    hipEventDestroy(start);
    hipEventDestroy(stop);


}

